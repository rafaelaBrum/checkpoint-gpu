
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

// using device-side mallocs that persist across kernel invocations
// based off of http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#allocation-persisting-kernel-launches
#define NUM_BLOCKS 1
__device__ int* tenptr[NUM_BLOCKS];

__global__ void alloc_and_set_ten()
{
    // Only the first thread in the block does the allocation
    // since we want only one allocation per block.
    if (threadIdx.x == 0) {
        tenptr[blockIdx.x] = (int*)malloc(sizeof(int));
        *tenptr[blockIdx.x] = 10;  // set the value
    }
    __syncthreads();
}

__global__ void add(int a, int b, int *c)
{
	*c = a+b+*tenptr[blockIdx.x];
}

__global__ void free_ten()
{
    // Free from the leader thread in each thread block
    if (threadIdx.x == 0)
        free(tenptr[blockIdx.x]);
}

int main(int argc, char **argv)
{
	// test
	int a = 2, b = 3, c;
	int *cuda_c = NULL;

	hipMalloc(&cuda_c, sizeof(int));
        sleep(10);
	alloc_and_set_ten<<<NUM_BLOCKS,1>>>();
	add<<<NUM_BLOCKS,1>>>(a, b, cuda_c);
    free_ten<<<NUM_BLOCKS,1>>>();
	hipMemcpy(&c, cuda_c, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(cuda_c);

	printf("%d + %d + 10 = %d\n", a, b, c);

	exit(EXIT_SUCCESS);
}
